#include "hip/hip_runtime.h"
#include "common/fmt.hpp"
#include "common/utils.hpp"

#define PRINT(...) LOG(info, std::string(fmt::format(__VA_ARGS__)))

#define NUM_COUNT  100000

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  float hostInput1[NUM_COUNT];
  float hostInput2[NUM_COUNT];
  float hostOutput[NUM_COUNT];
  float solution[NUM_COUNT];
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  for (int i = 0; i < NUM_COUNT; i++) {
    hostInput1[i] = NUM_COUNT / 1.0;
    hostInput2[i] = NUM_COUNT / 1.0;
    solution[i] = NUM_COUNT / 1.0 + NUM_COUNT / 1.0;
  }

  int size = NUM_COUNT * sizeof(float);
  hipMalloc((void **) &deviceInput1, size);
  hipMalloc((void **) &deviceInput2, size);
  hipMalloc((void **) &deviceOutput, size);

  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

  dim3 DimGrid(ceil(NUM_COUNT/1024.0),1,1);
  dim3 DimBlock(1024,1,1);

  timer_start("Start GPU Kernel");
  vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, NUM_COUNT);
  hipDeviceSynchronize();
  timer_stop();

  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  int err_cnt = 0;
  for (int i = 0; i < NUM_COUNT; i++) {
    if (hostOutput[i] != solution[i])
      err_cnt = 1;
  }

  if (err_cnt)
    PRINT("ERROR!");
  else
    PRINT("CORRECT!");

  return 0;
}